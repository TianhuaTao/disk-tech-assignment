#include "hip/hip_runtime.h"
/**
* Copyright 2016 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "fp16_emu.h"

// Usage
//   > ./RNN <flags>
// Command line flags
//   -seqLength<int>    : Specify sequence length
//   -numLayers<int>    : Specify number of layers
//   -hiddenSize<int>   : Specify hidden size
//   -miniBatch<int>    : Specify minibatch size
//   -dropout<float>    : Specify dropout probability
//   -bidirectional     : Switch to bidirectional instead of unidirectional RNN
//   -mode{0,1,2,3}     : Specify mode (ReLU, tanh, LSTM, GRU)
//   -persistent{0,1,2} : Specify recurrence algorithm (standard, persist dynamic, persist static)
//   -P{s,d,h}          : Specify data type precision (float, double, half)
//   -H                 : Display this help message
//
// Reference outputs (calculated on an M40 GPU)
// golden_1.txt (default case if you just run ./RNN)
// > ./RNN -seqLength20 -numLayers2 -hiddenSize512 -inputSize512 -miniBatch64 -mode0
// Forward: 1250 GFLOPS
// Backward: 1896 GFLOPS, (1299 GFLOPS), (3511 GFLOPS)
// y checksum 1.315793E+06     hy checksum 1.315212E+05
// dx checksum 6.676003E+01    dhx checksum 6.425050E+01
// dw checksum 1.453750E+09
//
// golden_2.txt
// > ./RNN -seqLength20 -numLayers2 -inputSize512 -hiddenSize512 -miniBatch64 -mode1
// Forward: 1225 GFLOPS
// Backward: 1910 GFLOPS, (1299 GFLOPS), (3601 GFLOPS)
// y checksum 6.319591E+05     hy checksum 6.319605E+04
// dx checksum 4.501830E+00    dhx checksum 4.489543E+00
// dw checksum 5.012598E+07
//
// golden_3.txt
// > ./RNN -seqLength20 -numLayers2 -inputSize512 -hiddenSize512 -miniBatch64 -mode2
// Forward: 2569 GFLOPS
// Backward: 2654 GFLOPS, (2071 GFLOPS), (3694 GFLOPS)
// y checksum 5.749536E+05     cy checksum 4.365091E+05     hy checksum 5.774818E+04
// dx checksum 3.842206E+02    dcx checksum 9.323785E+03    dhx checksum 1.182562E+01
// dw checksum 4.313461E+08
//
// golden_4.txt
// > ./RNN -seqLength20 -numLayers2 -inputSize512 -hiddenSize512 -miniBatch64 -mode3
// Forward: 2310 GFLOPS
// Backward: 2536 GFLOPS, (1955 GFLOPS), (3606 GFLOPS)
// y checksum 6.358978E+05     hy checksum 6.281680E+04
// dx checksum 6.296622E+00    dhx checksum 2.289960E+05
// dw checksum 5.397419E+07

// Templated functions to get hipdnnDataType_t from a templated type
template <typename T_ELEM> __inline__ hipdnnDataType_t getDataType();
template <> __inline__ hipdnnDataType_t getDataType<double>() { return HIPDNN_DATA_DOUBLE; }
template <> __inline__ hipdnnDataType_t getDataType<float>()  { return HIPDNN_DATA_FLOAT;  }
template <> __inline__ hipdnnDataType_t getDataType<half1>()  { return HIPDNN_DATA_HALF;   }

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

#define cudnnErrCheck(stat) { cudnnErrCheck_((stat), __FILE__, __LINE__); }
void cudnnErrCheck_(hipdnnStatus_t stat, const char *file, int line) {
    if (stat != HIPDNN_STATUS_SUCCESS) {
        fprintf(stderr, "cuDNN Error: %s %s %d\n", hipdnnGetErrorString(stat), file, line);
    }
}

// Kernel and launcher to initialize GPU data to some constant value
template <typename T_ELEM>
__global__
void initGPUData_ker(T_ELEM *data, int numElements, T_ELEM value) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numElements) {
        data[tid] = value;
    }
}

template <typename T_ELEM>
void initGPUData(T_ELEM *data, int numElements, T_ELEM value) {
    dim3 gridDim;
    dim3 blockDim;

    blockDim.x = 1024;
    gridDim.x  = (numElements + blockDim.x - 1) / blockDim.x;

    initGPUData_ker<<<gridDim, blockDim>>>(data, numElements, value);
}

// This function does all the work of setting up and running cuDNN's RNN functions with the given parameters.
// It also calculates performance results and checksums, printing them to the command line and saving them to
// result.txt for potential comparison to the files (golden_1.txt, etc).
template <typename T_ELEM>
void doTest(int seqLength,
            int numLayers,
            int hiddenSize,
            int inputSize,
            int miniBatch,
            float dropout,
            bool bidirectional,
            hipdnnRNNMode_t mode,
            hipdnnRNNAlgo_t algo) {
    FILE *fp;
    fp = fopen("result.txt", "w");

    // -------------------------
    // Create cudnn context
    // -------------------------
    hipdnnHandle_t cudnnHandle;
    cudnnErrCheck(hipdnnCreate(&cudnnHandle));

    // -------------------------
    // Set up inputs and outputs
    // -------------------------
    void *x;
    void *hx = NULL;
    void *cx = NULL;

    void *dx;
    void *dhx = NULL;
    void *dcx = NULL;

    void *y;
    void *hy = NULL;
    void *cy = NULL;

    void *dy;
    void *dhy = NULL;
    void *dcy = NULL;

    int dimIn[3];
    int dimOut[3];
    int dimHidden[3];
    int strideIn[3];
    int strideOut[3];
    int strideHidden[3];

    // Set up required dimensions for input, output, and hidden state tensors
    dimIn[0] = miniBatch;
    dimIn[1] = inputSize;
    dimIn[2] = 1;
    dimOut[0] = miniBatch;
    dimOut[1] = hiddenSize * (bidirectional ? 2 : 1);
    dimOut[2] = 1;
    dimHidden[0] = numLayers * (bidirectional ? 2 : 1);
    dimHidden[1] = miniBatch;
    dimHidden[2] = hiddenSize;

    strideIn[0] = dimIn[1] * dimIn[2];
    strideIn[1] = dimIn[2];
    strideIn[2] = 1;
    strideOut[0] = dimOut[1] * dimOut[2];
    strideOut[1] = dimOut[2];
    strideOut[2] = 1;
    strideHidden[0] = dimHidden[1] * dimHidden[2];
    strideHidden[1] = dimHidden[2];
    strideHidden[2] = 1;

    // Calculating total elements per each
    int inputTensorSize  = dimIn[0] * dimIn[1] * dimIn[2];
    int outputTensorSize = dimOut[0] * dimOut[1] * dimOut[2];
    int hiddenTensorSize = dimHidden[0] * dimHidden[1] * dimHidden[2];

    // Memory allocation. hx, cx, dhx, dcx, hy, cy, dhy and dcy can be NULL.
    cudaErrCheck(hipMalloc((void **)&x, seqLength * inputTensorSize * sizeof(T_ELEM)));
    cudaErrCheck(hipMalloc((void **)&y, seqLength * outputTensorSize * sizeof(T_ELEM)));
    cudaErrCheck(hipMalloc((void **)&dx, seqLength * inputTensorSize * sizeof(T_ELEM)));
    cudaErrCheck(hipMalloc((void **)&dy, seqLength * outputTensorSize * sizeof(T_ELEM)));

    cudaErrCheck(hipMalloc((void **)&hx, hiddenTensorSize * sizeof(T_ELEM)));
    cudaErrCheck(hipMalloc((void **)&cx, hiddenTensorSize * sizeof(T_ELEM)));
    cudaErrCheck(hipMalloc((void **)&hy, hiddenTensorSize * sizeof(T_ELEM)));
    cudaErrCheck(hipMalloc((void **)&cy, hiddenTensorSize * sizeof(T_ELEM)));
    cudaErrCheck(hipMalloc((void **)&dhx, hiddenTensorSize * sizeof(T_ELEM)));
    cudaErrCheck(hipMalloc((void **)&dcx, hiddenTensorSize * sizeof(T_ELEM)));
    cudaErrCheck(hipMalloc((void **)&dhy, hiddenTensorSize * sizeof(T_ELEM)));
    cudaErrCheck(hipMalloc((void **)&dcy, hiddenTensorSize * sizeof(T_ELEM)));

    // Set up tensor descriptors. x/y/dx/dy are arrays, one per time step.
    hipdnnTensorDescriptor_t *xDesc, *yDesc, *dxDesc, *dyDesc;
    hipdnnTensorDescriptor_t hxDesc, cxDesc;
    hipdnnTensorDescriptor_t hyDesc, cyDesc;
    hipdnnTensorDescriptor_t dhxDesc, dcxDesc;
    hipdnnTensorDescriptor_t dhyDesc, dcyDesc;

    xDesc  = (hipdnnTensorDescriptor_t *)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
    yDesc  = (hipdnnTensorDescriptor_t *)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
    dxDesc = (hipdnnTensorDescriptor_t *)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
    dyDesc = (hipdnnTensorDescriptor_t *)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));

    // In this example dimA[1] is constant across the whole sequence
    // This isn't required, all that is required is that it does not increase.
    for (int i = 0; i < seqLength; i++) {
        cudnnErrCheck(hipdnnCreateTensorDescriptor(&xDesc[i]));
        cudnnErrCheck(hipdnnCreateTensorDescriptor(&yDesc[i]));

        cudnnErrCheck(hipdnnCreateTensorDescriptor(&dxDesc[i]));
        cudnnErrCheck(hipdnnCreateTensorDescriptor(&dyDesc[i]));

        cudnnErrCheck(hipdnnSetTensorNdDescriptor(xDesc[i], getDataType<T_ELEM>(), 3, dimIn, strideIn));
        cudnnErrCheck(hipdnnSetTensorNdDescriptor(dxDesc[i], getDataType<T_ELEM>(), 3, dimIn, strideIn));

        cudnnErrCheck(hipdnnSetTensorNdDescriptor(yDesc[i], getDataType<T_ELEM>(), 3, dimOut, strideOut));
        cudnnErrCheck(hipdnnSetTensorNdDescriptor(dyDesc[i], getDataType<T_ELEM>(), 3, dimOut, strideOut));
    }

    cudnnErrCheck(hipdnnCreateTensorDescriptor(&hxDesc));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&cxDesc));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&hyDesc));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&cyDesc));

    cudnnErrCheck(hipdnnCreateTensorDescriptor(&dhxDesc));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&dcxDesc));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&dhyDesc));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&dcyDesc));

    cudnnErrCheck(hipdnnSetTensorNdDescriptor(hxDesc, getDataType<T_ELEM>(), 3, dimHidden, strideHidden));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(cxDesc, getDataType<T_ELEM>(), 3, dimHidden, strideHidden));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(hyDesc, getDataType<T_ELEM>(), 3, dimHidden, strideHidden));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(cyDesc, getDataType<T_ELEM>(), 3, dimHidden, strideHidden));

    cudnnErrCheck(hipdnnSetTensorNdDescriptor(dhxDesc, getDataType<T_ELEM>(), 3, dimHidden, strideHidden));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(dcxDesc, getDataType<T_ELEM>(), 3, dimHidden, strideHidden));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(dhyDesc, getDataType<T_ELEM>(), 3, dimHidden, strideHidden));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(dcyDesc, getDataType<T_ELEM>(), 3, dimHidden, strideHidden));

    // -------------------------
    // Set up the dropout descriptor (needed for the RNN descriptor)
    // -------------------------
    unsigned long long seed = 1337ull;  // Pick a seed.

    hipdnnDropoutDescriptor_t dropoutDesc;
    cudnnErrCheck(hipdnnCreateDropoutDescriptor(&dropoutDesc));

    // How much memory does dropout need for states?
    // These states are used to generate random numbers internally
    // and should not be freed until the RNN descriptor is no longer used
    size_t stateSize;
    void *states;
    cudnnErrCheck(hipdnnDropoutGetStatesSize(cudnnHandle, &stateSize));

    cudaErrCheck(hipMalloc(&states, stateSize));

    cudnnErrCheck(hipdnnSetDropoutDescriptor(dropoutDesc,
                                            cudnnHandle,
                                            dropout,
                                            states,
                                            stateSize,
                                            seed));

    // -------------------------
    // Set up the RNN descriptor
    // -------------------------
    hipdnnRNNDescriptor_t rnnDesc;

    cudnnErrCheck(hipdnnCreateRNNDescriptor(&rnnDesc));

    cudnnErrCheck(hipdnnSetRNNDescriptor_v6(cudnnHandle,
                                           rnnDesc,
                                           hiddenSize,
                                           numLayers,
                                           dropoutDesc,
                                           HIPDNN_LINEAR_INPUT,  // We can also skip the input matrix transformation
                                           bidirectional ? HIPDNN_BIDIRECTIONAL : HIPDNN_UNIDIRECTIONAL,
                                           mode,
                                           algo,  // Can be changed to use persistent RNNs on Pascal+ GPUs.
                                           getDataType<T_ELEM>()));

    // -------------------------
    // Set up parameters
    // -------------------------
    // This needs to be done after the rnn descriptor is set as otherwise
    // we don't know how many parameters we have to allocate
    void *w;
    void *dw;

    hipdnnFilterDescriptor_t wDesc, dwDesc;

    cudnnErrCheck(hipdnnCreateFilterDescriptor(&wDesc));
    cudnnErrCheck(hipdnnCreateFilterDescriptor(&dwDesc));

    size_t weightsSize;
    cudnnErrCheck(hipdnnGetRNNParamsSize(cudnnHandle, rnnDesc, xDesc[0], &weightsSize, getDataType<T_ELEM>()));

    int dimW[3];
    dimW[0] = weightsSize / sizeof(T_ELEM);
    dimW[1] = 1;
    dimW[2] = 1;

    cudnnErrCheck(hipdnnSetFilterNdDescriptor(wDesc, getDataType<T_ELEM>(), HIPDNN_TENSOR_NCHW, 3, dimW));
    cudnnErrCheck(hipdnnSetFilterNdDescriptor(dwDesc, getDataType<T_ELEM>(), HIPDNN_TENSOR_NCHW, 3, dimW));

    cudaErrCheck(hipMalloc((void **)&w, weightsSize));
    cudaErrCheck(hipMalloc((void **)&dw, weightsSize));

    // -------------------------
    // Set up work space and reserved memory
    // -------------------------
    void *workspace;
    void *reserveSpace;

    size_t workSize;
    size_t reserveSize;

    // Need for every pass
    cudnnErrCheck(hipdnnGetRNNWorkspaceSize(cudnnHandle, rnnDesc, seqLength, xDesc, &workSize));
    // Only needed in training, shouldn't be touched between passes.
    cudnnErrCheck(hipdnnGetRNNTrainingReserveSize(cudnnHandle, rnnDesc, seqLength, xDesc, &reserveSize));

    cudaErrCheck(hipMalloc((void **)&workspace, workSize));
    cudaErrCheck(hipMalloc((void **)&reserveSpace, reserveSize));

    // *********************************************************************************************************
    // Initialise weights and inputs
    // *********************************************************************************************************
    // We initialise to something simple.
    // Matrices are initialised to 1 / matrixSize, biases to 1, data is 1.

    //Initialize inputs
    initGPUData<T_ELEM>((T_ELEM *)x, seqLength * inputTensorSize, 1.0);
    if (hx != NULL) initGPUData<T_ELEM>((T_ELEM *)hx, hiddenTensorSize, 1.0);
    if (cx != NULL) initGPUData<T_ELEM>((T_ELEM *)cx, hiddenTensorSize, 1.0);

    initGPUData<T_ELEM>((T_ELEM *)dy, seqLength * outputTensorSize, 1.0);
    if (dhy != NULL) initGPUData<T_ELEM>((T_ELEM *)dhy, hiddenTensorSize, 1.0);
    if (dcy != NULL) initGPUData<T_ELEM>((T_ELEM *)dcy, hiddenTensorSize, 1.0);


    // Initialize Weights
    int numLinearLayers = 0;
    if (mode == HIPDNN_RNN_RELU || mode == HIPDNN_RNN_TANH) {
        numLinearLayers = 2;
    } else if (mode == HIPDNN_LSTM) {
        numLinearLayers = 8;
    } else if (mode == HIPDNN_GRU) {
        numLinearLayers = 6;
    }

    for (int layer = 0; layer < numLayers * (bidirectional ? 2 : 1); layer++) {
        for (int linLayerID = 0; linLayerID < numLinearLayers; linLayerID++) {
            hipdnnDataType_t dataType;
            hipdnnTensorFormat_t format;
            int nbDims;
            int filterDimA[3];

            //Initialize layer weights
            hipdnnFilterDescriptor_t linLayerMatDesc;
            T_ELEM *linLayerMat;

            cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerMatDesc));
            cudnnErrCheck(hipdnnGetRNNLinLayerMatrixParams(cudnnHandle,
                                                          rnnDesc,
                                                          layer,
                                                          xDesc[0],
                                                          wDesc,
                                                          w,
                                                          linLayerID,
                                                          linLayerMatDesc,
                                                          (void **)&linLayerMat));

            cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerMatDesc, 3, &dataType, &format, &nbDims, filterDimA));

            initGPUData<T_ELEM>(linLayerMat, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.0 / (filterDimA[0] * filterDimA[1] * filterDimA[2]));

            cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerMatDesc));

            //Initialize layer bias
            hipdnnFilterDescriptor_t linLayerBiasDesc;
            T_ELEM *linLayerBias;

            cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerBiasDesc));
            cudnnErrCheck(hipdnnGetRNNLinLayerBiasParams(cudnnHandle,
                                                        rnnDesc,
                                                        layer,
                                                        xDesc[0],
                                                        wDesc,
                                                        w,
                                                        linLayerID,
                                                        linLayerBiasDesc,
                                                        (void **)&linLayerBias));

            cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerBiasDesc, 3, &dataType, &format, &nbDims, filterDimA));

            initGPUData<T_ELEM>(linLayerBias, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.0);

            cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerBiasDesc));
        }
    }

    // *********************************************************************************************************
    // Dynamic persistent RNN plan (if using this algo)
    // *********************************************************************************************************
    hipdnnPersistentRNNPlan_t rnnPlan;
    if (algo == HIPDNN_RNN_ALGO_PERSIST_DYNAMIC) {
        // Note: This step is expensive. Once completed the plan can be reused so long as the descriptor
        //       minibatch or datatype don't change.
        cudnnErrCheck(hipdnnCreatePersistentRNNPlan(rnnDesc, miniBatch, getDataType<T_ELEM>(), &rnnPlan));
        // Tell calls using this descriptor which plan to use.
        cudnnErrCheck(hipdnnSetPersistentRNNPlan(rnnDesc, rnnPlan));
    }

    // *********************************************************************************************************
    // At this point all of the setup is done. We now need to pass through the RNN.
    // *********************************************************************************************************
    cudaErrCheck(hipDeviceSynchronize());

    hipEvent_t start, stop;
    float timeForward, timeBackward1, timeBackward2;
    cudaErrCheck(hipEventCreate(&start));
    cudaErrCheck(hipEventCreate(&stop));

    cudaErrCheck(hipEventRecord(start));

    // If we're not training we use this instead
    // cudnnErrCheck(hipdnnRNNForwardInference(cudnnHandle,
    // rnnDesc,
    // seqLength,
    // xDesc,
    // x,
    // hxDesc,
    // hx,
    // cxDesc,
    // cx,
    // wDesc,
    // w,
    // yDesc,
    // y,
    // hyDesc,
    // hy,
    // cyDesc,
    // cy,
    // workspace,
    // workSize));

    cudnnErrCheck(hipdnnRNNForwardTraining(cudnnHandle,
                                          rnnDesc,
                                          seqLength,
                                          xDesc,
                                          x,
                                          hxDesc,
                                          hx,
                                          cxDesc,
                                          cx,
                                          wDesc,
                                          w,
                                          yDesc,
                                          y,
                                          hyDesc,
                                          hy,
                                          cyDesc,
                                          cy,
                                          workspace,
                                          workSize,
                                          reserveSpace,
                                          reserveSize));

    cudaErrCheck(hipEventRecord(stop));
    cudaErrCheck(hipEventSynchronize(stop));
    cudaErrCheck(hipEventElapsedTime(&timeForward, start, stop));

    cudaErrCheck(hipEventRecord(start));

    cudnnErrCheck(hipdnnRNNBackwardData(cudnnHandle,
                                       rnnDesc,
                                       seqLength,
                                       yDesc,
                                       y,
                                       dyDesc,
                                       dy,
                                       dhyDesc,
                                       dhy,
                                       dcyDesc,
                                       dcy,
                                       wDesc,
                                       w,
                                       hxDesc,
                                       hx,
                                       cxDesc,
                                       cx,
                                       dxDesc,
                                       dx,
                                       dhxDesc,
                                       dhx,
                                       dcxDesc,
                                       dcx,
                                       workspace,
                                       workSize,
                                       reserveSpace,
                                       reserveSize));

    cudaErrCheck(hipEventRecord(stop));
    cudaErrCheck(hipEventSynchronize(stop));
    cudaErrCheck(hipEventElapsedTime(&timeBackward1, start, stop));

    cudaErrCheck(hipEventRecord(start));

    // hipdnnRNNBackwardWeights adds to the data in dw.
    cudaErrCheck(hipMemset(dw, 0, weightsSize));

    cudnnErrCheck(hipdnnRNNBackwardWeights(cudnnHandle,
                                          rnnDesc,
                                          seqLength,
                                          xDesc,
                                          x,
                                          hxDesc,
                                          hx,
                                          yDesc,
                                          y,
                                          workspace,
                                          workSize,
                                          dwDesc,
                                          dw,
                                          reserveSpace,
                                          reserveSize));

    cudaErrCheck(hipEventRecord(stop));

    cudaErrCheck(hipEventSynchronize(stop));
    cudaErrCheck(hipEventElapsedTime(&timeBackward2, start, stop));

    int numMats = 0;

    if (mode == HIPDNN_RNN_RELU || mode == HIPDNN_RNN_TANH) {
        numMats = 2;
    } else if (mode == HIPDNN_LSTM) {
        numMats = 8;
    } else if (mode == HIPDNN_GRU) {
        numMats = 6;
    }

    // Calculate FLOPS
    printf("Forward: %3.0f GFLOPS\n", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeForward));
    printf("Backward: %3.0f GFLOPS, ", numMats * 4ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * (timeBackward1 + timeBackward2)));
    printf("(%3.0f GFLOPS), ", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward1));
    printf("(%3.0f GFLOPS)\n", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward2));

    // Save FLOPS to file
    fprintf(fp, "Forward: %3.0f GFLOPS\n", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeForward));
    fprintf(fp, "Backward: %3.0f GFLOPS, ", numMats * 4ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * (timeBackward1 + timeBackward2)));
    fprintf(fp, "(%3.0f GFLOPS), ", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward1));
    fprintf(fp, "(%3.0f GFLOPS)\n", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward2));

    // Make double-sure everything is finished before we copy for result checking.
    hipDeviceSynchronize();

    // *********************************************************************************************************
    // Print checksums.
    // *********************************************************************************************************
    int biDirScale = (bidirectional ? 2 : 1);

    if (true) {
        T_ELEM *testOutputy;
        T_ELEM *testOutputhy;
        T_ELEM *testOutputcy;

        testOutputy = (T_ELEM *)malloc(seqLength * outputTensorSize * sizeof(T_ELEM));
        testOutputhy = (T_ELEM *)malloc(hiddenTensorSize * sizeof(T_ELEM));
        testOutputcy = (T_ELEM *)malloc(hiddenTensorSize * sizeof(T_ELEM));

        cudaErrCheck(hipMemcpy(testOutputy, y, seqLength * outputTensorSize * sizeof(T_ELEM), hipMemcpyDeviceToHost));
        if (hy != NULL) {
            cudaErrCheck(hipMemcpy(testOutputhy, hy, hiddenTensorSize * sizeof(T_ELEM), hipMemcpyDeviceToHost));
        }
        if (cy != NULL && mode == HIPDNN_LSTM) {
            cudaErrCheck(hipMemcpy(testOutputcy, cy, hiddenTensorSize * sizeof(T_ELEM), hipMemcpyDeviceToHost));
        }

        double checksumy = 0.f;
        double checksumhy = 0.f;
        double checksumcy = 0.f;

        for (int m = 0; m < miniBatch; m++) {
            double localSumi = 0;
            double localSumh = 0;
            double localSumc = 0;

            for (int j = 0; j < seqLength; j++) {
                for (int i = 0; i < hiddenSize * biDirScale; i++) {
                    localSumi += (double) testOutputy[j * miniBatch * hiddenSize * biDirScale + m * hiddenSize * biDirScale + i];
                }
            }
            for (int j = 0; j < numLayers * biDirScale; j++) {
                for (int i = 0; i < hiddenSize; i++) {
                    if (hy != NULL) {
                        localSumh += (double) testOutputhy[j * hiddenSize * miniBatch + m * hiddenSize + i];
                    }
                    if ((cy != NULL) && (mode == HIPDNN_LSTM)) {
                        localSumc += (double) testOutputcy[j * hiddenSize * miniBatch + m * hiddenSize + i];
                    }
                }
            }

            checksumy += localSumi;
            checksumhy += localSumh;
            checksumcy += localSumc;
        }

        printf("y checksum %E     ", checksumy);
        fprintf(fp, "y checksum %E     ", checksumy);
        if (mode == HIPDNN_LSTM) {
            printf("cy checksum %E     ", checksumcy);
            fprintf(fp, "cy checksum %E     ", checksumcy);
        }
        printf("hy checksum %E\n", checksumhy);
        fprintf(fp, "hy checksum %E\n", checksumhy);

        free(testOutputy);
        free(testOutputcy);
        free(testOutputhy);
    }

    if (true) {
        T_ELEM *testOutputdx;
        T_ELEM *testOutputdhx;
        T_ELEM *testOutputdcx;

        testOutputdx = (T_ELEM *)malloc(seqLength * inputTensorSize * sizeof(T_ELEM));
        testOutputdhx = (T_ELEM *)malloc(hiddenTensorSize * sizeof(T_ELEM));
        testOutputdcx = (T_ELEM *)malloc(hiddenTensorSize * sizeof(T_ELEM));

        cudaErrCheck(hipMemcpy(testOutputdx, dx, seqLength * inputTensorSize * sizeof(T_ELEM), hipMemcpyDeviceToHost));
        if (dhx != NULL) {
            cudaErrCheck(hipMemcpy(testOutputdhx, dhx, hiddenTensorSize * sizeof(T_ELEM), hipMemcpyDeviceToHost));
        }
        if ((dcx != NULL) && (mode == HIPDNN_LSTM)) {
            cudaErrCheck(hipMemcpy(testOutputdcx, dcx, hiddenTensorSize * sizeof(T_ELEM), hipMemcpyDeviceToHost));
        }

        double checksumdx = 0.f;
        double checksumdhx = 0.f;
        double checksumdcx = 0.f;

        for (int m = 0; m < miniBatch; m++) {
            double localSumdx = 0;
            double localSumdhx = 0;
            double localSumdcx = 0;

            for (int j = 0; j < seqLength; j++) {
                for (int i = 0; i < inputSize; i++) {
                    localSumdx += (double) testOutputdx[j * miniBatch * inputSize + m * inputSize + i];
                }
            }

            for (int j = 0; j < numLayers * biDirScale; j++) {
                for (int i = 0; i < hiddenSize; i++) {
                    localSumdhx += (double) testOutputdhx[j * hiddenSize * miniBatch + m * hiddenSize + i];
                    if (mode == HIPDNN_LSTM) {
                        localSumdcx += (double) testOutputdcx[j * hiddenSize * miniBatch + m * hiddenSize + i];
                    }
                }
            }

            checksumdx += localSumdx;
            checksumdhx += localSumdhx;
            checksumdcx += localSumdcx;
        }

        printf("dx checksum %E    ", checksumdx);
        fprintf(fp, "dx checksum %E    ", checksumdx);
        if (mode == HIPDNN_LSTM) {
            printf("dcx checksum %E    ", checksumdcx);
            fprintf(fp, "dcx checksum %E    ", checksumdcx);
        }
        printf("dhx checksum %E\n", checksumdhx);
        fprintf(fp, "dhx checksum %E\n", checksumdhx);

        free(testOutputdx);
        free(testOutputdhx);
        free(testOutputdcx);
    }

    if (true) {
        T_ELEM *testOutputdw;
        testOutputdw = (T_ELEM *)malloc(weightsSize);

        cudaErrCheck(hipMemcpy(testOutputdw, dw, weightsSize, hipMemcpyDeviceToHost));

        double checksumdw = 0.;

        for (int i = 0; i < weightsSize / sizeof(T_ELEM); i++) {
            checksumdw += (double) testOutputdw[i];
        }

        printf("dw checksum %E\n", checksumdw);
        fprintf(fp, "dw checksum %E\n", checksumdw);

        free(testOutputdw);
    }

    //Free all previously allocated memory, destroy all created cudnn descriptors
    if (algo == HIPDNN_RNN_ALGO_PERSIST_DYNAMIC) {
        hipdnnDestroyPersistentRNNPlan(rnnPlan);
    }

    hipFree(x);
    hipFree(hx);
    hipFree(cx);
    hipFree(y);
    hipFree(hy);
    hipFree(cy);
    hipFree(dx);
    hipFree(dhx);
    hipFree(dcx);
    hipFree(dy);
    hipFree(dhy);
    hipFree(dcy);
    hipFree(workspace);
    hipFree(reserveSpace);
    hipFree(w);
    hipFree(dw);
    hipFree(states);

    for (int i = 0; i < seqLength; i++) {
        hipdnnDestroyTensorDescriptor(xDesc[i]);
        hipdnnDestroyTensorDescriptor(yDesc[i]);
        hipdnnDestroyTensorDescriptor(dxDesc[i]);
        hipdnnDestroyTensorDescriptor(dyDesc[i]);
    }

    free(xDesc);
    free(yDesc);
    free(dxDesc);
    free(dyDesc);

    hipdnnDestroyTensorDescriptor(hxDesc);
    hipdnnDestroyTensorDescriptor(cxDesc);
    hipdnnDestroyTensorDescriptor(hyDesc);
    hipdnnDestroyTensorDescriptor(cyDesc);
    hipdnnDestroyTensorDescriptor(dhxDesc);
    hipdnnDestroyTensorDescriptor(dcxDesc);
    hipdnnDestroyTensorDescriptor(dhyDesc);
    hipdnnDestroyTensorDescriptor(dcyDesc);

    hipdnnDestroyDropoutDescriptor(dropoutDesc);
    hipdnnDestroyRNNDescriptor(rnnDesc);
    hipdnnDestroyFilterDescriptor(wDesc);
    hipdnnDestroyFilterDescriptor(dwDesc);

    hipdnnDestroy(cudnnHandle);
    printf("Output saved to result.txt\n");
    fclose(fp);
}

// Reads command line arguments and stores them in the proper variables
int parse_args(int argc,
               char *argv[],
               int &seqLength,
               int &numLayers,
               int &hiddenSize,
               int &inputSize,
               int &miniBatch,
               float &dropout,
               bool &bidirectional,
               hipdnnRNNMode_t &mode,
               hipdnnRNNAlgo_t &algo,
               hipdnnDataType_t &dataType) {
    argc -= 1;
    argv++;
    while (argc) {
        if (argv[0][0] == '-') {
            if (strncmp(argv[0] + 1, "seqLength", strlen("seqLength")) == 0) {
                seqLength = atoi(argv[0] + 1 + strlen("seqLength"));
            } else if (strncmp(argv[0] + 1, "numLayers", strlen("numLayers")) == 0) {
                numLayers = atoi(argv[0] + 1 + strlen("numLayers"));
            } else if (strncmp(argv[0] + 1, "hiddenSize", strlen("hiddenSize")) == 0) {
                hiddenSize = atoi(argv[0] + 1 + strlen("hiddenSize"));
                inputSize = hiddenSize; // For now we fix inputSize = hiddenSize
            } else if (strncmp(argv[0] + 1, "miniBatch", strlen("miniBatch")) == 0) {
                miniBatch = atoi(argv[0] + 1 + strlen("miniBatch"));
            } else if (strncmp(argv[0] + 1, "dropout", strlen("dropout")) == 0) {
                char *p = argv[0] + 1 + strlen("dropout");
                sscanf(p, "%f", &dropout);
            } else if (strncmp(argv[0] + 1, "bidirectional", strlen("bidirectional")) == 0) {
                bidirectional = true;
            } else if (strncmp(argv[0] + 1, "mode", strlen("mode")) == 0) {
                mode = (hipdnnRNNMode_t) atoi(argv[0] + 1 + strlen("mode"));
            } else if (strncmp(argv[0] + 1, "algo", strlen("algo")) == 0) {
                algo = (hipdnnRNNAlgo_t) atoi(argv[0] + 1 + strlen("algo"));
            } else if (strncmp(argv[0] + 1, "Ps", strlen("Ps")) == 0) {
                dataType = HIPDNN_DATA_FLOAT;
            } else if (strncmp(argv[0] + 1, "Pd", strlen("Pd")) == 0) {
                dataType = HIPDNN_DATA_DOUBLE;
            } else if (strncmp(argv[0] + 1, "Ph", strlen("Ph")) == 0) {
                dataType = HIPDNN_DATA_HALF;
            } else if (argv[0][1] == 'H') {
                printf("Usage\n");
                printf("  > ./RNN <flags>\n");
                printf("Command line flags\n");
                printf("  -seqLength<int>    : Specify sequence length\n");
                printf("  -numLayers<int>    : Specify number of layers\n");
                printf("  -hiddenSize<int>   : Specify hidden size\n");
                printf("  -miniBatch<int>    : Specify minibatch size\n");
                printf("  -dropout<float>    : Specify dropout probability\n");
                printf("  -bidirectional     : Switch to bidirectional instead of unidirectional RNN\n");
                printf("  -mode{0,1,2,3}     : Specify mode (ReLU, tanh, LSTM, GRU)\n");
                printf("  -persistent{0,1,2} : Specify recurrence algorithm (standard, persist dynamic, persist static)\n");
                printf("  -P{s,d,h}          : Specify data type precision (float, double, half)\n");
                printf("  -H                 : Display this help message\n");
                return 1;
            } else {
                printf("Improper command line flag! See \"./RNN -H\" for proper usage\n");
                return 1;
            }
        } else {
            printf("Improper command line flag! See \"./RNN -H\" for proper usage\n");
            return 1;
        }
        argc -= 1;
        argv++;
    }

    return 0;
}

// Checks whether the given parameters are supported by cuDNN, and prints them to the command line
int print_args(int seqLength,
               int numLayers,
               int hiddenSize,
               int inputSize,
               int miniBatch,
               float dropout,
               bool bidirectional,
               hipdnnRNNMode_t mode,
               hipdnnRNNAlgo_t algo,
               hipdnnDataType_t dataType) {
    printf("seqLength  = %d\nnumLayers  = %d\nhiddenSize = %d\n", seqLength, numLayers, hiddenSize);
    printf("inputSize  = %d\nminiBatch  = %d\ndropout    = %.6f\n", inputSize, miniBatch, dropout);

    if (bidirectional) {
        printf("direction  = HIPDNN_BIDIRECTIONAL\n");
    } else {
        printf("direction  = HIPDNN_UNIDIRECTIONAL\n");
    }

    if (mode == HIPDNN_RNN_RELU) {
        printf("mode       = HIPDNN_RNN_RELU\n");
    } else if (mode == HIPDNN_RNN_TANH) {
        printf("mode       = HIPDNN_RNN_TANH\n");
    } else if (mode == HIPDNN_LSTM) {
        printf("mode       = HIPDNN_LSTM\n");
    } else if (mode == HIPDNN_GRU) {
        printf("mode       = HIPDNN_GRU\n");
    }

    if (algo == HIPDNN_RNN_ALGO_STANDARD) {
        printf("algo       = HIPDNN_RNN_ALGO_STANDARD\n");
    } else {
        // Persistent RNNs are only supported on Pascal+ GPUs.
        int device;
        struct hipDeviceProp_t devProp;
        hipGetDevice(&device);
        hipGetDeviceProperties(&devProp, device);
        if (devProp.major < 6) {
            printf("!!! ERROR: Persistent RNNs are only supported on Pascal+ GPUs\n");
            return 1;
        }
        if (algo == HIPDNN_RNN_ALGO_PERSIST_STATIC) {
            printf("algo       = HIPDNN_RNN_ALGO_PERSIST_STATIC\n");
        } else if (algo == HIPDNN_RNN_ALGO_PERSIST_DYNAMIC) {
            printf("algo       = HIPDNN_RNN_ALGO_PERSIST_DYNAMIC\n");
        }
    }

    if (dataType == HIPDNN_DATA_FLOAT) {
        printf("precision  = HIPDNN_DATA_FLOAT\n");
    } else if (dataType == HIPDNN_DATA_DOUBLE) {
        printf("precision  = HIPDNN_DATA_DOUBLE\n");
        if (algo == HIPDNN_RNN_ALGO_PERSIST_STATIC) {
            printf("!!! ERROR: Double precision is disabled for PERSIST_STATIC algorithm\n");
            return 1;
        }
    } else if (dataType == HIPDNN_DATA_HALF) {
        printf("precision  = HIPDNN_DATA_HALF\n");
    }

    printf("\n");
    return 0;
}

int main(int argc, char *argv[]) {
    // Default case is same as golden_1.txt
    int seqLength = 20;
    int numLayers = 2;
    int hiddenSize = 512;
    int inputSize = hiddenSize; // For now we fix inputSize = hiddenSize
    int miniBatch = 64;
    float dropout = 0;
    bool bidirectional = false;
    hipdnnRNNMode_t mode = HIPDNN_RNN_RELU;
    hipdnnRNNAlgo_t algo = HIPDNN_RNN_ALGO_STANDARD;
    hipdnnDataType_t dataType = HIPDNN_DATA_FLOAT;

    // Read in command line flags
    bool error = parse_args(argc,
                            argv,
                            seqLength,
                            numLayers,
                            hiddenSize,
                            inputSize,
                            miniBatch,
                            dropout,
                            bidirectional,
                            mode,
                            algo,
                            dataType);

    if (error) {
        return 0;
    }

    // Check and print arguments before performing test
    error = print_args(seqLength,
                       numLayers,
                       hiddenSize,
                       inputSize,
                       miniBatch,
                       dropout,
                       bidirectional,
                       mode,
                       algo,
                       dataType);

    if (error) {
        return 0;
    }

    // Perform test depending on precision
    if (dataType == HIPDNN_DATA_FLOAT) {
        doTest<float>(seqLength, numLayers, hiddenSize, inputSize, miniBatch, dropout, bidirectional, mode, algo);
    } else if (dataType == HIPDNN_DATA_DOUBLE) {
        doTest<double>(seqLength, numLayers, hiddenSize, inputSize, miniBatch, dropout, bidirectional, mode, algo);
    } else if (dataType == HIPDNN_DATA_HALF) {
        doTest<half1>(seqLength, numLayers, hiddenSize, inputSize, miniBatch, dropout, bidirectional, mode, algo);
    }

    return 0;
}
